#include "hip/hip_runtime.h"
#include"vec_add_device/vec_add_device.h"

__global__ void vec_add_kernel(const uint64_t n, float *c, const float *a, const float *b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
 
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

void vec_add_device(hipStream_t stream, const uint64_t n, float *d_c, const float *d_a, const float *d_b) {
    uint64_t blockSize = 1;
    uint64_t gridSize = n;
    vec_add_kernel<<<gridSize, blockSize, 0, stream>>>(n, d_c, d_a, d_b);
}